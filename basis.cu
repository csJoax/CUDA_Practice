
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int num)
{
    int i = threadIdx.x;
    if (i < num)
    {
        c[i] = b[i] + a[i];
    }
}

int main(int argc, char const *argv[])
{
    // init data
    const int num = 10;
    int a[num], b[num], c[num];
    int *a_gpu, *b_gpu, *c_gpu;
    for (auto i = 0; i < num; i++)
    {
        a[i] = i;
        b[i] = i * i;
    }
    hipMalloc((void **)&a_gpu, sizeof(a));
    hipMalloc((void **)&b_gpu, sizeof(b));
    hipMalloc((void **)&c_gpu, sizeof(c));

    // copy data
    hipMemcpy(a_gpu, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, sizeof(b), hipMemcpyHostToDevice);

    // do
    add<<<1, num>>>(a_gpu, b_gpu, c_gpu, num);

    hipMemcpy(c, c_gpu, sizeof(c), hipMemcpyDeviceToHost);

    // viz
    for (size_t i = 0; i < num; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
